#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <unistd.h>
#include <stdint.h>

#define POP 300
#define LEN 30
#define MUT 0.1
#define REC 0.5
#define END 10000
#define SUMTAG 150
#define PRODTAG 3600

int gene[POP][LEN];
int value[POP][LEN];
int seed[POP][LEN];
void init_pop();
double evaluate(int n);
void run();
void display(int tournaments, int n);
void get_result(int idx);
double score[POP];

double random_double(){
  double d;
  d = (double)(rand() % 10) / 10;
  return d; 
}

__device__ unsigned int get_rand(int range, int* seed){
  *seed ^= (*seed << 13);
  *seed ^= (*seed >> 17);
  *seed ^= (*seed << 5);
  return *seed % range;
}
         
__global__ void comput_kernel(double* score, int* gene, int* value){
  int offset = blockIdx.x * LEN;
  //evaluate
  uint64_t prod = 1;
  uint32_t sum = 0;
  for(int i = 0; i < LEN; ++i)
    if(gene[offset + i] == 0)
      sum += value[offset + i];
    else
      prod *= value[offset + i];

  double scaled_sum_error = (double)(sum - (double)SUMTAG) / (double)SUMTAG;
  double scaled_prod_error = (double)(prod - (double)PRODTAG) / (double)PRODTAG;
  
  if(scaled_sum_error < 0.0) 
    scaled_sum_error *= -1;
  
  if(scaled_prod_error < 0.0)
    scaled_prod_error *= -1;
  
  score[blockIdx.x] = scaled_sum_error + scaled_prod_error;
  
}

__global__ void find_min_score(double *score, double* min_score, int *min_idx){
  int low_idx = -1;
  double low = 0;
  
  __shared__ double shared_score[POP];

  for(int i = threadIdx.x; i < POP; i += blockDim.x)
    shared_score[i] = score[i];
  
  __syncthreads();
  if(threadIdx.x != 0)
    return;

  for(int i = 0; i < POP; ++i)
    if(shared_score[i] < low || low_idx == -1){
      low = score[i];
      low_idx = i;
    }

  *min_idx = low_idx;
  *min_score = low;
}


__global__ void mutate_kernel(int* gene, int *min_idx, int* seed){
  if(blockIdx.x == *min_idx)
    return;

  int offset = blockIdx.x * LEN;
  int min_offset = *min_idx * LEN;
  int reg_seed = seed[blockIdx.x * LEN + threadIdx.x];

  if(get_rand(100, &reg_seed) < (REC / 100))
    gene[offset + threadIdx.x] = gene[min_offset + threadIdx.x];
  if(get_rand(100, &reg_seed) < (MUT / 100))
    gene[offset + threadIdx.x] = 1 - gene[offset + threadIdx.x];

  seed[blockIdx.x * LEN + threadIdx.x] = reg_seed;
}

void run(){
  init_pop();
  int low_idx = -1;
  double low = 0;   
  int tournamentNo;
  
  int* gene_d;
  int* value_d;
  double* score_d;
  double* min_score;
  int* min_idx;
  int* seed_d;

  hipMalloc((void**)&gene_d, sizeof(int) * POP * LEN); 
  hipMalloc((void**)&value_d, sizeof(int) * POP * LEN); 
  hipMalloc((void**)&score_d, sizeof(double) * POP); 
  hipMalloc((void**)&seed_d, sizeof(int) * POP * LEN); 
  hipMalloc((void**)&min_score, sizeof(double)); 
  hipMalloc((void**)&min_idx, sizeof(int)); 
  

  hipMemcpy(gene_d, gene, sizeof(int) * POP * LEN, hipMemcpyHostToDevice);
  hipMemcpy(value_d, value, sizeof(int) * POP * LEN, hipMemcpyHostToDevice);
  hipMemcpy(score_d, score, sizeof(double) * POP, hipMemcpyHostToDevice);
  hipMemcpy(seed_d, seed, sizeof(int) * POP * LEN, hipMemcpyHostToDevice);

  dim3 dimGrid(POP, 1);
  dim3 dimBlock(1, 1);

  for(tournamentNo = 0; tournamentNo < END; tournamentNo++){
    comput_kernel<<<dimGrid, dimBlock>>>(score_d, gene_d, value_d);
    find_min_score<<<1, 128>>>(score_d, min_score, min_idx); 
    mutate_kernel<<<dimGrid, LEN>>>(gene_d, min_idx, seed_d);
  }
  
  hipMemcpy(gene, gene_d, sizeof(int) * POP * LEN, hipMemcpyDeviceToHost);
  hipMemcpy(score, score_d, sizeof(double) * POP, hipMemcpyDeviceToHost);
  
  low_idx = -1;
  low = 0;
  for(int i =0; i < POP; ++i)
    if((low_idx == -1 || score[i] < low) && score[i] != -1){
      low = score[i];
      low_idx = i;
    }
  
  if(low_idx != -1){
    //printf("%f %f\n", low, evaluate(low_idx));
    get_result(low_idx);
    display(tournamentNo, low_idx);
  }
}

void get_result(int idx){
  unsigned long long prod, sum;
  prod = 1;
  sum = 0;

  for(int i = 0; i < LEN; ++i){
    if(gene[idx][i] == 1)
      prod *= value[idx][i];
    else
      sum += value[idx][i];
  }

  printf("sum :%llu  prod: %llu\n", sum, prod);
}

void display(int tournaments, int n){
  printf("=========================================================================\n");
  printf("After %d tournaments, Solution sum pile (should be %d) cards are : \n", tournaments, SUMTAG);
  for(int i = 0; i < LEN; i++){
      if(gene[n][i] == 0){
          printf("%d ", value[n][i]);
      } 
  }
  printf("\n");
  printf("Solution product pile (should be %d) cards are : \n", PRODTAG);
  for(int i = 0; i < LEN; i++){
      if(gene[n][i] == 1){
          printf("%d ", value[n][i]);
      } 
  }
  
  for(int i = 0; i < LEN; i++)
      assert(gene[n][i] == 1 || gene[n][i] == 0);
  printf("\n=========================================================================\n")
}


double evaluate(int n){
  unsigned long long sum = 0, prod = 1;
  double scaled_sum_error, scaled_prod_error, combined_error;
  for(int i = 0; i < LEN; i++){
      if(gene[n][i] == 0){
          sum += value[n][i];
      }
      else{
         prod *= value[n][i];
      }
  }

  scaled_sum_error = (double)(sum - (double)SUMTAG) / (double)SUMTAG;
  if(scaled_sum_error < 0.0) scaled_sum_error *= -1;
  scaled_prod_error = (double)(prod - (double)PRODTAG) / (double)PRODTAG;
  if(scaled_prod_error < 0.0) scaled_prod_error *= -1;
  combined_error = scaled_sum_error + scaled_prod_error;
  return combined_error;
}


void init_pop(){
  for(int i = 0; i < POP; i++){
      for(int j = 0; j < LEN; j++){
          if(random_double() < 0.5){
              gene[i][j] = 0;
          }
          else{
              gene[i][j] = 1;
          }
      }
    score[i] = -1;
  }
  
  for(int i = 0; i < POP; i++){
      for(int j = 0; j < LEN; j++){
          value[i][j] = rand() % 9 + 1;
      }
  }

  for(int i = 0; i < POP; ++i)
    for(int j = 0; j < LEN; ++j)
    seed[i][j] = rand() % 10000000;
}

int main(){
  srand(getpid());  
  run();
  return 0;
}

